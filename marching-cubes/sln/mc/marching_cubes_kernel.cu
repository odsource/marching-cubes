#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime_api.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
//#include "Marching_Cubes.hpp"
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
//#include <vector>
#include "marching_cubes_kernel.cuh"

#define BLOCK_SIZE 512

// Rendering variables
float xmax = 5.0f;
float xmin = -5.0f;
int numPoints = 2;
int func = 0;

///////////////////////////////////////////////////////////////////////////////
// Marching cubes table data												 //
///////////////////////////////////////////////////////////////////////////////

// Bit mask for edges
__device__
int edgeTable[256] = {
0x0  , 0x109, 0x203, 0x30a, 0x406, 0x50f, 0x605, 0x70c,
0x80c, 0x905, 0xa0f, 0xb06, 0xc0a, 0xd03, 0xe09, 0xf00,
0x190, 0x99 , 0x393, 0x29a, 0x596, 0x49f, 0x795, 0x69c,
0x99c, 0x895, 0xb9f, 0xa96, 0xd9a, 0xc93, 0xf99, 0xe90,
0x230, 0x339, 0x33 , 0x13a, 0x636, 0x73f, 0x435, 0x53c,
0xa3c, 0xb35, 0x83f, 0x936, 0xe3a, 0xf33, 0xc39, 0xd30,
0x3a0, 0x2a9, 0x1a3, 0xaa , 0x7a6, 0x6af, 0x5a5, 0x4ac,
0xbac, 0xaa5, 0x9af, 0x8a6, 0xfaa, 0xea3, 0xda9, 0xca0,
0x460, 0x569, 0x663, 0x76a, 0x66 , 0x16f, 0x265, 0x36c,
0xc6c, 0xd65, 0xe6f, 0xf66, 0x86a, 0x963, 0xa69, 0xb60,
0x5f0, 0x4f9, 0x7f3, 0x6fa, 0x1f6, 0xff , 0x3f5, 0x2fc,
0xdfc, 0xcf5, 0xfff, 0xef6, 0x9fa, 0x8f3, 0xbf9, 0xaf0,
0x650, 0x759, 0x453, 0x55a, 0x256, 0x35f, 0x55 , 0x15c,
0xe5c, 0xf55, 0xc5f, 0xd56, 0xa5a, 0xb53, 0x859, 0x950,
0x7c0, 0x6c9, 0x5c3, 0x4ca, 0x3c6, 0x2cf, 0x1c5, 0xcc ,
0xfcc, 0xec5, 0xdcf, 0xcc6, 0xbca, 0xac3, 0x9c9, 0x8c0,
0x8c0, 0x9c9, 0xac3, 0xbca, 0xcc6, 0xdcf, 0xec5, 0xfcc,
0xcc , 0x1c5, 0x2cf, 0x3c6, 0x4ca, 0x5c3, 0x6c9, 0x7c0,
0x950, 0x859, 0xb53, 0xa5a, 0xd56, 0xc5f, 0xf55, 0xe5c,
0x15c, 0x55 , 0x35f, 0x256, 0x55a, 0x453, 0x759, 0x650,
0xaf0, 0xbf9, 0x8f3, 0x9fa, 0xef6, 0xfff, 0xcf5, 0xdfc,
0x2fc, 0x3f5, 0xff , 0x1f6, 0x6fa, 0x7f3, 0x4f9, 0x5f0,
0xb60, 0xa69, 0x963, 0x86a, 0xf66, 0xe6f, 0xd65, 0xc6c,
0x36c, 0x265, 0x16f, 0x66 , 0x76a, 0x663, 0x569, 0x460,
0xca0, 0xda9, 0xea3, 0xfaa, 0x8a6, 0x9af, 0xaa5, 0xbac,
0x4ac, 0x5a5, 0x6af, 0x7a6, 0xaa , 0x1a3, 0x2a9, 0x3a0,
0xd30, 0xc39, 0xf33, 0xe3a, 0x936, 0x83f, 0xb35, 0xa3c,
0x53c, 0x435, 0x73f, 0x636, 0x13a, 0x33 , 0x339, 0x230,
0xe90, 0xf99, 0xc93, 0xd9a, 0xa96, 0xb9f, 0x895, 0x99c,
0x69c, 0x795, 0x49f, 0x596, 0x29a, 0x393, 0x99 , 0x190,
0xf00, 0xe09, 0xd03, 0xc0a, 0xb06, 0xa0f, 0x905, 0x80c,
0x70c, 0x605, 0x50f, 0x406, 0x30a, 0x203, 0x109, 0x0 };

// Edge mapping
__device__
int edge_map[12][2] = {
    {0,1}, {1,2}, {2,3}, {3,0},
    {4,5}, {5,6}, {6,7}, {7,4},
    {0,4}, {1,5}, {2,6}, {3,7},
};

// Edges which will be used for triangle vertices
__device__
int triTable[256][16] =
{ {-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 8, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 1, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 8, 3, 9, 8, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 8, 3, 1, 2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{9, 2, 10, 0, 2, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{2, 8, 3, 2, 10, 8, 10, 9, 8, -1, -1, -1, -1, -1, -1, -1},
{3, 11, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 11, 2, 8, 11, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 9, 0, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 11, 2, 1, 9, 11, 9, 8, 11, -1, -1, -1, -1, -1, -1, -1},
{3, 10, 1, 11, 10, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 10, 1, 0, 8, 10, 8, 11, 10, -1, -1, -1, -1, -1, -1, -1},
{3, 9, 0, 3, 11, 9, 11, 10, 9, -1, -1, -1, -1, -1, -1, -1},
{9, 8, 10, 10, 8, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{4, 7, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{4, 3, 0, 7, 3, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 1, 9, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{4, 1, 9, 4, 7, 1, 7, 3, 1, -1, -1, -1, -1, -1, -1, -1},
{1, 2, 10, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{3, 4, 7, 3, 0, 4, 1, 2, 10, -1, -1, -1, -1, -1, -1, -1},
{9, 2, 10, 9, 0, 2, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1},
{2, 10, 9, 2, 9, 7, 2, 7, 3, 7, 9, 4, -1, -1, -1, -1},
{8, 4, 7, 3, 11, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{11, 4, 7, 11, 2, 4, 2, 0, 4, -1, -1, -1, -1, -1, -1, -1},
{9, 0, 1, 8, 4, 7, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1},
{4, 7, 11, 9, 4, 11, 9, 11, 2, 9, 2, 1, -1, -1, -1, -1},
{3, 10, 1, 3, 11, 10, 7, 8, 4, -1, -1, -1, -1, -1, -1, -1},
{1, 11, 10, 1, 4, 11, 1, 0, 4, 7, 11, 4, -1, -1, -1, -1},
{4, 7, 8, 9, 0, 11, 9, 11, 10, 11, 0, 3, -1, -1, -1, -1},
{4, 7, 11, 4, 11, 9, 9, 11, 10, -1, -1, -1, -1, -1, -1, -1},
{9, 5, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{9, 5, 4, 0, 8, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 5, 4, 1, 5, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{8, 5, 4, 8, 3, 5, 3, 1, 5, -1, -1, -1, -1, -1, -1, -1},
{1, 2, 10, 9, 5, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{3, 0, 8, 1, 2, 10, 4, 9, 5, -1, -1, -1, -1, -1, -1, -1},
{5, 2, 10, 5, 4, 2, 4, 0, 2, -1, -1, -1, -1, -1, -1, -1},
{2, 10, 5, 3, 2, 5, 3, 5, 4, 3, 4, 8, -1, -1, -1, -1},
{9, 5, 4, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 11, 2, 0, 8, 11, 4, 9, 5, -1, -1, -1, -1, -1, -1, -1},
{0, 5, 4, 0, 1, 5, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1},
{2, 1, 5, 2, 5, 8, 2, 8, 11, 4, 8, 5, -1, -1, -1, -1},
{10, 3, 11, 10, 1, 3, 9, 5, 4, -1, -1, -1, -1, -1, -1, -1},
{4, 9, 5, 0, 8, 1, 8, 10, 1, 8, 11, 10, -1, -1, -1, -1},
{5, 4, 0, 5, 0, 11, 5, 11, 10, 11, 0, 3, -1, -1, -1, -1},
{5, 4, 8, 5, 8, 10, 10, 8, 11, -1, -1, -1, -1, -1, -1, -1},
{9, 7, 8, 5, 7, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{9, 3, 0, 9, 5, 3, 5, 7, 3, -1, -1, -1, -1, -1, -1, -1},
{0, 7, 8, 0, 1, 7, 1, 5, 7, -1, -1, -1, -1, -1, -1, -1},
{1, 5, 3, 3, 5, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{9, 7, 8, 9, 5, 7, 10, 1, 2, -1, -1, -1, -1, -1, -1, -1},
{10, 1, 2, 9, 5, 0, 5, 3, 0, 5, 7, 3, -1, -1, -1, -1},
{8, 0, 2, 8, 2, 5, 8, 5, 7, 10, 5, 2, -1, -1, -1, -1},
{2, 10, 5, 2, 5, 3, 3, 5, 7, -1, -1, -1, -1, -1, -1, -1},
{7, 9, 5, 7, 8, 9, 3, 11, 2, -1, -1, -1, -1, -1, -1, -1},
{9, 5, 7, 9, 7, 2, 9, 2, 0, 2, 7, 11, -1, -1, -1, -1},
{2, 3, 11, 0, 1, 8, 1, 7, 8, 1, 5, 7, -1, -1, -1, -1},
{11, 2, 1, 11, 1, 7, 7, 1, 5, -1, -1, -1, -1, -1, -1, -1},
{9, 5, 8, 8, 5, 7, 10, 1, 3, 10, 3, 11, -1, -1, -1, -1},
{5, 7, 0, 5, 0, 9, 7, 11, 0, 1, 0, 10, 11, 10, 0, -1},
{11, 10, 0, 11, 0, 3, 10, 5, 0, 8, 0, 7, 5, 7, 0, -1},
{11, 10, 5, 7, 11, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{10, 6, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 8, 3, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{9, 0, 1, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 8, 3, 1, 9, 8, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1},
{1, 6, 5, 2, 6, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 6, 5, 1, 2, 6, 3, 0, 8, -1, -1, -1, -1, -1, -1, -1},
{9, 6, 5, 9, 0, 6, 0, 2, 6, -1, -1, -1, -1, -1, -1, -1},
{5, 9, 8, 5, 8, 2, 5, 2, 6, 3, 2, 8, -1, -1, -1, -1},
{2, 3, 11, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{11, 0, 8, 11, 2, 0, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1},
{0, 1, 9, 2, 3, 11, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1},
{5, 10, 6, 1, 9, 2, 9, 11, 2, 9, 8, 11, -1, -1, -1, -1},
{6, 3, 11, 6, 5, 3, 5, 1, 3, -1, -1, -1, -1, -1, -1, -1},
{0, 8, 11, 0, 11, 5, 0, 5, 1, 5, 11, 6, -1, -1, -1, -1},
{3, 11, 6, 0, 3, 6, 0, 6, 5, 0, 5, 9, -1, -1, -1, -1},
{6, 5, 9, 6, 9, 11, 11, 9, 8, -1, -1, -1, -1, -1, -1, -1},
{5, 10, 6, 4, 7, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{4, 3, 0, 4, 7, 3, 6, 5, 10, -1, -1, -1, -1, -1, -1, -1},
{1, 9, 0, 5, 10, 6, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1},
{10, 6, 5, 1, 9, 7, 1, 7, 3, 7, 9, 4, -1, -1, -1, -1},
{6, 1, 2, 6, 5, 1, 4, 7, 8, -1, -1, -1, -1, -1, -1, -1},
{1, 2, 5, 5, 2, 6, 3, 0, 4, 3, 4, 7, -1, -1, -1, -1},
{8, 4, 7, 9, 0, 5, 0, 6, 5, 0, 2, 6, -1, -1, -1, -1},
{7, 3, 9, 7, 9, 4, 3, 2, 9, 5, 9, 6, 2, 6, 9, -1},
{3, 11, 2, 7, 8, 4, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1},
{5, 10, 6, 4, 7, 2, 4, 2, 0, 2, 7, 11, -1, -1, -1, -1},
{0, 1, 9, 4, 7, 8, 2, 3, 11, 5, 10, 6, -1, -1, -1, -1},
{9, 2, 1, 9, 11, 2, 9, 4, 11, 7, 11, 4, 5, 10, 6, -1},
{8, 4, 7, 3, 11, 5, 3, 5, 1, 5, 11, 6, -1, -1, -1, -1},
{5, 1, 11, 5, 11, 6, 1, 0, 11, 7, 11, 4, 0, 4, 11, -1},
{0, 5, 9, 0, 6, 5, 0, 3, 6, 11, 6, 3, 8, 4, 7, -1},
{6, 5, 9, 6, 9, 11, 4, 7, 9, 7, 11, 9, -1, -1, -1, -1},
{10, 4, 9, 6, 4, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{4, 10, 6, 4, 9, 10, 0, 8, 3, -1, -1, -1, -1, -1, -1, -1},
{10, 0, 1, 10, 6, 0, 6, 4, 0, -1, -1, -1, -1, -1, -1, -1},
{8, 3, 1, 8, 1, 6, 8, 6, 4, 6, 1, 10, -1, -1, -1, -1},
{1, 4, 9, 1, 2, 4, 2, 6, 4, -1, -1, -1, -1, -1, -1, -1},
{3, 0, 8, 1, 2, 9, 2, 4, 9, 2, 6, 4, -1, -1, -1, -1},
{0, 2, 4, 4, 2, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{8, 3, 2, 8, 2, 4, 4, 2, 6, -1, -1, -1, -1, -1, -1, -1},
{10, 4, 9, 10, 6, 4, 11, 2, 3, -1, -1, -1, -1, -1, -1, -1},
{0, 8, 2, 2, 8, 11, 4, 9, 10, 4, 10, 6, -1, -1, -1, -1},
{3, 11, 2, 0, 1, 6, 0, 6, 4, 6, 1, 10, -1, -1, -1, -1},
{6, 4, 1, 6, 1, 10, 4, 8, 1, 2, 1, 11, 8, 11, 1, -1},
{9, 6, 4, 9, 3, 6, 9, 1, 3, 11, 6, 3, -1, -1, -1, -1},
{8, 11, 1, 8, 1, 0, 11, 6, 1, 9, 1, 4, 6, 4, 1, -1},
{3, 11, 6, 3, 6, 0, 0, 6, 4, -1, -1, -1, -1, -1, -1, -1},
{6, 4, 8, 11, 6, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{7, 10, 6, 7, 8, 10, 8, 9, 10, -1, -1, -1, -1, -1, -1, -1},
{0, 7, 3, 0, 10, 7, 0, 9, 10, 6, 7, 10, -1, -1, -1, -1},
{10, 6, 7, 1, 10, 7, 1, 7, 8, 1, 8, 0, -1, -1, -1, -1},
{10, 6, 7, 10, 7, 1, 1, 7, 3, -1, -1, -1, -1, -1, -1, -1},
{1, 2, 6, 1, 6, 8, 1, 8, 9, 8, 6, 7, -1, -1, -1, -1},
{2, 6, 9, 2, 9, 1, 6, 7, 9, 0, 9, 3, 7, 3, 9, -1},
{7, 8, 0, 7, 0, 6, 6, 0, 2, -1, -1, -1, -1, -1, -1, -1},
{7, 3, 2, 6, 7, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{2, 3, 11, 10, 6, 8, 10, 8, 9, 8, 6, 7, -1, -1, -1, -1},
{2, 0, 7, 2, 7, 11, 0, 9, 7, 6, 7, 10, 9, 10, 7, -1},
{1, 8, 0, 1, 7, 8, 1, 10, 7, 6, 7, 10, 2, 3, 11, -1},
{11, 2, 1, 11, 1, 7, 10, 6, 1, 6, 7, 1, -1, -1, -1, -1},
{8, 9, 6, 8, 6, 7, 9, 1, 6, 11, 6, 3, 1, 3, 6, -1},
{0, 9, 1, 11, 6, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{7, 8, 0, 7, 0, 6, 3, 11, 0, 11, 6, 0, -1, -1, -1, -1},
{7, 11, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{7, 6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{3, 0, 8, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 1, 9, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{8, 1, 9, 8, 3, 1, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1},
{10, 1, 2, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 2, 10, 3, 0, 8, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1},
{2, 9, 0, 2, 10, 9, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1},
{6, 11, 7, 2, 10, 3, 10, 8, 3, 10, 9, 8, -1, -1, -1, -1},
{7, 2, 3, 6, 2, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{7, 0, 8, 7, 6, 0, 6, 2, 0, -1, -1, -1, -1, -1, -1, -1},
{2, 7, 6, 2, 3, 7, 0, 1, 9, -1, -1, -1, -1, -1, -1, -1},
{1, 6, 2, 1, 8, 6, 1, 9, 8, 8, 7, 6, -1, -1, -1, -1},
{10, 7, 6, 10, 1, 7, 1, 3, 7, -1, -1, -1, -1, -1, -1, -1},
{10, 7, 6, 1, 7, 10, 1, 8, 7, 1, 0, 8, -1, -1, -1, -1},
{0, 3, 7, 0, 7, 10, 0, 10, 9, 6, 10, 7, -1, -1, -1, -1},
{7, 6, 10, 7, 10, 8, 8, 10, 9, -1, -1, -1, -1, -1, -1, -1},
{6, 8, 4, 11, 8, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{3, 6, 11, 3, 0, 6, 0, 4, 6, -1, -1, -1, -1, -1, -1, -1},
{8, 6, 11, 8, 4, 6, 9, 0, 1, -1, -1, -1, -1, -1, -1, -1},
{9, 4, 6, 9, 6, 3, 9, 3, 1, 11, 3, 6, -1, -1, -1, -1},
{6, 8, 4, 6, 11, 8, 2, 10, 1, -1, -1, -1, -1, -1, -1, -1},
{1, 2, 10, 3, 0, 11, 0, 6, 11, 0, 4, 6, -1, -1, -1, -1},
{4, 11, 8, 4, 6, 11, 0, 2, 9, 2, 10, 9, -1, -1, -1, -1},
{10, 9, 3, 10, 3, 2, 9, 4, 3, 11, 3, 6, 4, 6, 3, -1},
{8, 2, 3, 8, 4, 2, 4, 6, 2, -1, -1, -1, -1, -1, -1, -1},
{0, 4, 2, 4, 6, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 9, 0, 2, 3, 4, 2, 4, 6, 4, 3, 8, -1, -1, -1, -1},
{1, 9, 4, 1, 4, 2, 2, 4, 6, -1, -1, -1, -1, -1, -1, -1},
{8, 1, 3, 8, 6, 1, 8, 4, 6, 6, 10, 1, -1, -1, -1, -1},
{10, 1, 0, 10, 0, 6, 6, 0, 4, -1, -1, -1, -1, -1, -1, -1},
{4, 6, 3, 4, 3, 8, 6, 10, 3, 0, 3, 9, 10, 9, 3, -1},
{10, 9, 4, 6, 10, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{4, 9, 5, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 8, 3, 4, 9, 5, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1},
{5, 0, 1, 5, 4, 0, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1},
{11, 7, 6, 8, 3, 4, 3, 5, 4, 3, 1, 5, -1, -1, -1, -1},
{9, 5, 4, 10, 1, 2, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1},
{6, 11, 7, 1, 2, 10, 0, 8, 3, 4, 9, 5, -1, -1, -1, -1},
{7, 6, 11, 5, 4, 10, 4, 2, 10, 4, 0, 2, -1, -1, -1, -1},
{3, 4, 8, 3, 5, 4, 3, 2, 5, 10, 5, 2, 11, 7, 6, -1},
{7, 2, 3, 7, 6, 2, 5, 4, 9, -1, -1, -1, -1, -1, -1, -1},
{9, 5, 4, 0, 8, 6, 0, 6, 2, 6, 8, 7, -1, -1, -1, -1},
{3, 6, 2, 3, 7, 6, 1, 5, 0, 5, 4, 0, -1, -1, -1, -1},
{6, 2, 8, 6, 8, 7, 2, 1, 8, 4, 8, 5, 1, 5, 8, -1},
{9, 5, 4, 10, 1, 6, 1, 7, 6, 1, 3, 7, -1, -1, -1, -1},
{1, 6, 10, 1, 7, 6, 1, 0, 7, 8, 7, 0, 9, 5, 4, -1},
{4, 0, 10, 4, 10, 5, 0, 3, 10, 6, 10, 7, 3, 7, 10, -1},
{7, 6, 10, 7, 10, 8, 5, 4, 10, 4, 8, 10, -1, -1, -1, -1},
{6, 9, 5, 6, 11, 9, 11, 8, 9, -1, -1, -1, -1, -1, -1, -1},
{3, 6, 11, 0, 6, 3, 0, 5, 6, 0, 9, 5, -1, -1, -1, -1},
{0, 11, 8, 0, 5, 11, 0, 1, 5, 5, 6, 11, -1, -1, -1, -1},
{6, 11, 3, 6, 3, 5, 5, 3, 1, -1, -1, -1, -1, -1, -1, -1},
{1, 2, 10, 9, 5, 11, 9, 11, 8, 11, 5, 6, -1, -1, -1, -1},
{0, 11, 3, 0, 6, 11, 0, 9, 6, 5, 6, 9, 1, 2, 10, -1},
{11, 8, 5, 11, 5, 6, 8, 0, 5, 10, 5, 2, 0, 2, 5, -1},
{6, 11, 3, 6, 3, 5, 2, 10, 3, 10, 5, 3, -1, -1, -1, -1},
{5, 8, 9, 5, 2, 8, 5, 6, 2, 3, 8, 2, -1, -1, -1, -1},
{9, 5, 6, 9, 6, 0, 0, 6, 2, -1, -1, -1, -1, -1, -1, -1},
{1, 5, 8, 1, 8, 0, 5, 6, 8, 3, 8, 2, 6, 2, 8, -1},
{1, 5, 6, 2, 1, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 3, 6, 1, 6, 10, 3, 8, 6, 5, 6, 9, 8, 9, 6, -1},
{10, 1, 0, 10, 0, 6, 9, 5, 0, 5, 6, 0, -1, -1, -1, -1},
{0, 3, 8, 5, 6, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{10, 5, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{11, 5, 10, 7, 5, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{11, 5, 10, 11, 7, 5, 8, 3, 0, -1, -1, -1, -1, -1, -1, -1},
{5, 11, 7, 5, 10, 11, 1, 9, 0, -1, -1, -1, -1, -1, -1, -1},
{10, 7, 5, 10, 11, 7, 9, 8, 1, 8, 3, 1, -1, -1, -1, -1},
{11, 1, 2, 11, 7, 1, 7, 5, 1, -1, -1, -1, -1, -1, -1, -1},
{0, 8, 3, 1, 2, 7, 1, 7, 5, 7, 2, 11, -1, -1, -1, -1},
{9, 7, 5, 9, 2, 7, 9, 0, 2, 2, 11, 7, -1, -1, -1, -1},
{7, 5, 2, 7, 2, 11, 5, 9, 2, 3, 2, 8, 9, 8, 2, -1},
{2, 5, 10, 2, 3, 5, 3, 7, 5, -1, -1, -1, -1, -1, -1, -1},
{8, 2, 0, 8, 5, 2, 8, 7, 5, 10, 2, 5, -1, -1, -1, -1},
{9, 0, 1, 5, 10, 3, 5, 3, 7, 3, 10, 2, -1, -1, -1, -1},
{9, 8, 2, 9, 2, 1, 8, 7, 2, 10, 2, 5, 7, 5, 2, -1},
{1, 3, 5, 3, 7, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 8, 7, 0, 7, 1, 1, 7, 5, -1, -1, -1, -1, -1, -1, -1},
{9, 0, 3, 9, 3, 5, 5, 3, 7, -1, -1, -1, -1, -1, -1, -1},
{9, 8, 7, 5, 9, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{5, 8, 4, 5, 10, 8, 10, 11, 8, -1, -1, -1, -1, -1, -1, -1},
{5, 0, 4, 5, 11, 0, 5, 10, 11, 11, 3, 0, -1, -1, -1, -1},
{0, 1, 9, 8, 4, 10, 8, 10, 11, 10, 4, 5, -1, -1, -1, -1},
{10, 11, 4, 10, 4, 5, 11, 3, 4, 9, 4, 1, 3, 1, 4, -1},
{2, 5, 1, 2, 8, 5, 2, 11, 8, 4, 5, 8, -1, -1, -1, -1},
{0, 4, 11, 0, 11, 3, 4, 5, 11, 2, 11, 1, 5, 1, 11, -1},
{0, 2, 5, 0, 5, 9, 2, 11, 5, 4, 5, 8, 11, 8, 5, -1},
{9, 4, 5, 2, 11, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{2, 5, 10, 3, 5, 2, 3, 4, 5, 3, 8, 4, -1, -1, -1, -1},
{5, 10, 2, 5, 2, 4, 4, 2, 0, -1, -1, -1, -1, -1, -1, -1},
{3, 10, 2, 3, 5, 10, 3, 8, 5, 4, 5, 8, 0, 1, 9, -1},
{5, 10, 2, 5, 2, 4, 1, 9, 2, 9, 4, 2, -1, -1, -1, -1},
{8, 4, 5, 8, 5, 3, 3, 5, 1, -1, -1, -1, -1, -1, -1, -1},
{0, 4, 5, 1, 0, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{8, 4, 5, 8, 5, 3, 9, 0, 5, 0, 3, 5, -1, -1, -1, -1},
{9, 4, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{4, 11, 7, 4, 9, 11, 9, 10, 11, -1, -1, -1, -1, -1, -1, -1},
{0, 8, 3, 4, 9, 7, 9, 11, 7, 9, 10, 11, -1, -1, -1, -1},
{1, 10, 11, 1, 11, 4, 1, 4, 0, 7, 4, 11, -1, -1, -1, -1},
{3, 1, 4, 3, 4, 8, 1, 10, 4, 7, 4, 11, 10, 11, 4, -1},
{4, 11, 7, 9, 11, 4, 9, 2, 11, 9, 1, 2, -1, -1, -1, -1},
{9, 7, 4, 9, 11, 7, 9, 1, 11, 2, 11, 1, 0, 8, 3, -1},
{11, 7, 4, 11, 4, 2, 2, 4, 0, -1, -1, -1, -1, -1, -1, -1},
{11, 7, 4, 11, 4, 2, 8, 3, 4, 3, 2, 4, -1, -1, -1, -1},
{2, 9, 10, 2, 7, 9, 2, 3, 7, 7, 4, 9, -1, -1, -1, -1},
{9, 10, 7, 9, 7, 4, 10, 2, 7, 8, 7, 0, 2, 0, 7, -1},
{3, 7, 10, 3, 10, 2, 7, 4, 10, 1, 10, 0, 4, 0, 10, -1},
{1, 10, 2, 8, 7, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{4, 9, 1, 4, 1, 7, 7, 1, 3, -1, -1, -1, -1, -1, -1, -1},
{4, 9, 1, 4, 1, 7, 0, 8, 1, 8, 7, 1, -1, -1, -1, -1},
{4, 0, 3, 7, 4, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{4, 8, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{9, 10, 8, 10, 11, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{3, 0, 9, 3, 9, 11, 11, 9, 10, -1, -1, -1, -1, -1, -1, -1},
{0, 1, 10, 0, 10, 8, 8, 10, 11, -1, -1, -1, -1, -1, -1, -1},
{3, 1, 10, 11, 3, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 2, 11, 1, 11, 9, 9, 11, 8, -1, -1, -1, -1, -1, -1, -1},
{3, 0, 9, 3, 9, 11, 1, 2, 9, 2, 11, 9, -1, -1, -1, -1},
{0, 2, 11, 8, 0, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{3, 2, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{2, 3, 8, 2, 8, 10, 10, 8, 9, -1, -1, -1, -1, -1, -1, -1},
{9, 10, 2, 0, 9, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{2, 3, 8, 2, 8, 10, 0, 1, 8, 1, 10, 8, -1, -1, -1, -1},
{1, 10, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{1, 3, 8, 9, 1, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 9, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{0, 3, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
{-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1} };

// number of vertices for each case above
__device__
int numVertsTable[256] =
{
    0,
    3,
    3,
    6,
    3,
    6,
    6,
    9,
    3,
    6,
    6,
    9,
    6,
    9,
    9,
    6,
    3,
    6,
    6,
    9,
    6,
    9,
    9,
    12,
    6,
    9,
    9,
    12,
    9,
    12,
    12,
    9,
    3,
    6,
    6,
    9,
    6,
    9,
    9,
    12,
    6,
    9,
    9,
    12,
    9,
    12,
    12,
    9,
    6,
    9,
    9,
    6,
    9,
    12,
    12,
    9,
    9,
    12,
    12,
    9,
    12,
    15,
    15,
    6,
    3,
    6,
    6,
    9,
    6,
    9,
    9,
    12,
    6,
    9,
    9,
    12,
    9,
    12,
    12,
    9,
    6,
    9,
    9,
    12,
    9,
    12,
    12,
    15,
    9,
    12,
    12,
    15,
    12,
    15,
    15,
    12,
    6,
    9,
    9,
    12,
    9,
    12,
    6,
    9,
    9,
    12,
    12,
    15,
    12,
    15,
    9,
    6,
    9,
    12,
    12,
    9,
    12,
    15,
    9,
    6,
    12,
    15,
    15,
    12,
    15,
    6,
    12,
    3,
    3,
    6,
    6,
    9,
    6,
    9,
    9,
    12,
    6,
    9,
    9,
    12,
    9,
    12,
    12,
    9,
    6,
    9,
    9,
    12,
    9,
    12,
    12,
    15,
    9,
    6,
    12,
    9,
    12,
    9,
    15,
    6,
    6,
    9,
    9,
    12,
    9,
    12,
    12,
    15,
    9,
    12,
    12,
    15,
    12,
    15,
    15,
    12,
    9,
    12,
    12,
    9,
    12,
    15,
    15,
    12,
    12,
    9,
    15,
    6,
    15,
    12,
    6,
    3,
    6,
    9,
    9,
    12,
    9,
    12,
    12,
    15,
    9,
    12,
    12,
    15,
    6,
    9,
    9,
    6,
    9,
    12,
    12,
    15,
    12,
    15,
    15,
    6,
    12,
    9,
    15,
    12,
    9,
    6,
    12,
    3,
    9,
    12,
    12,
    15,
    12,
    15,
    9,
    12,
    12,
    15,
    15,
    6,
    9,
    12,
    6,
    3,
    6,
    9,
    9,
    6,
    9,
    12,
    6,
    3,
    9,
    6,
    12,
    3,
    6,
    3,
    3,
    0,
};

__global__ void points_kernel(float4* points, int size, int func);
__global__ void kernel3D(float4* points, float4* geom, int size);

void run_cuda_kernel(GLuint* vao, GLuint* vbo1, GLuint* vbo2)
{
    glBindVertexArray(vao[0]);
    // Map OpenGL buffer object for writing from CUDA
    float4* dev_points;
    float4* dev_geometry;

    // Map OpenGL buffers to CUDA
    if (cudaGLMapBufferObject((void**)&dev_points, vbo1[1]) != hipSuccess) 
    {
        printf("Points buffer could not be mapped to CUDA!\r\n");
    }
    
    if (cudaGLMapBufferObject((void**)&dev_geometry, vbo1[2]) != hipSuccess)
    {
        printf("Geometry buffer could not be mapped to CUDA!\r\n");
    }
    
    // Choose a block size and a grid size
    const unsigned int threadsPerBlock = BLOCK_SIZE;
    const unsigned int maxBlocks = 50;
    unsigned int blocks;

    // Execute CUDA kernels

    blocks = min(maxBlocks,
        (int)ceil(numPoints * numPoints * numPoints / (float)threadsPerBlock));
    printf("\tNumber of CUDA blocks: %d\r\n", blocks);
    printf("\tNumber of threads per block: %d\r\n", threadsPerBlock);
    // Check for containment of vertices
    points_kernel << <blocks, threadsPerBlock >> >
        (dev_points, numPoints, func);
    // Obtain the triangles from the data table
    kernel3D << <blocks, threadsPerBlock >> >
        (dev_points, dev_geometry, numPoints);

    // Unmap buffer objects from CUDA
    if (cudaGLUnmapBufferObject(vbo1[1]) != hipSuccess)
    {
        printf("Could not unmap vbo from CUDA!\r\n");
    }
    
    if (cudaGLUnmapBufferObject(vbo1[2]) != hipSuccess)
    {
        printf("Could not unmap vbo from CUDA!\r\n");
    }
    
    // Second VAO

    glBindVertexArray(vao[1]);
    // Map OpenGL buffer object for writing from CUDA
    float4* dev_points2;
    float4* dev_geometry2;

    // Map OpenGL buffers to CUDA
    if (cudaGLMapBufferObject((void**)&dev_points2, vbo2[1]) != hipSuccess)
    {
        printf("Points buffer could not be mapped to CUDA!\r\n");
    }

    if (cudaGLMapBufferObject((void**)&dev_geometry2, vbo2[2]) != hipSuccess)
    {
        printf("Geometry buffer could not be mapped to CUDA!\r\n");
    }

    // Execute CUDA kernels

    blocks = min(maxBlocks,
        (int)ceil(numPoints * numPoints * numPoints / (float)threadsPerBlock));

    // Check for containment of vertices
    points_kernel << <blocks, threadsPerBlock >> >
        (dev_points2, numPoints, func);
    // Obtain the triangles from the data table
    kernel3D << <blocks, threadsPerBlock >> >
        (dev_points2, dev_geometry2, numPoints);

    // Unmap buffer objects from CUDA
    if (cudaGLUnmapBufferObject(vbo2[1]) != hipSuccess)
    {
        printf("Could not unmap vbo from CUDA!\r\n");
    }

    if (cudaGLUnmapBufferObject(vbo2[2]) != hipSuccess)
    {
        printf("Could not unmap vbo from CUDA!\r\n");
    }
    
}

__device__ __host__
int density_func(float4& point, int func)
{
    float fun; int flag;
    switch (func) {

    case 0:
        fun = point.x * point.x + point.y * point.y + point.z * point.z;
        flag = (fun < 9);
        break;

    case 1:
        fun = point.x * point.x / 5.0 + point.y * point.y / 3.0
            - point.z * point.z / 7.0;
        flag = (fun < 5);
        break;

    case 2:
        fun = point.x * point.x / 10.0 - point.y * point.y / 3.0
            - point.z / 2.0;
        flag = (fun < 0);
        break;
    }

    return flag;
}

// This kernel checks whether each point lies within the desired surface.
__global__
void points_kernel(float4* points, int size, int func)
{
    unsigned int globalID = blockIdx.x * blockDim.x + threadIdx.x;

    for (int k = globalID; k < size * size * size; k += gridDim.x * blockDim.x) {
        float4 pt = points[k];
        points[k].w = density_func(pt, func);
    }
}

// This kernel classifies each cube in the grid.
__global__
void kernel3D(float4* points, float4* geom, int size)
{
    // Get unique thread ID, this is the point ID
    unsigned int globalID = blockIdx.x * blockDim.x + threadIdx.x;

    for (int id = globalID; id < size * size * size; id += gridDim.x * blockDim.x) {

        // Transform point ID to cube ID
        int j = (int)((int)floor((double)(id / size)) % size);
        int k = (int)floor((double)(id / (size * size)));
        int idx = id - j + k - 2 * k * size;

        if (idx < (size - 1) * (size - 1) * (size - 1)) {

            // Get the vertices of this cube
            float4 verts[8];
            verts[0] = points[id];
            verts[1] = points[id + 1];
            verts[2] = points[id + size + 1];
            verts[3] = points[id + size];

            verts[4] = points[id + size * size];
            verts[5] = points[id + size * size + 1];
            verts[6] = points[id + size * size + size + 1];
            verts[7] = points[id + size * size + size];

            // Obtain the type/index of this cube
            int type = 0;
            for (int l = 0; l < 8; l++) {
                type += verts[l].w * pow((double)2, (double)l);
            }

            // Get the configuration for this type of cube from the table
            // and generate the triangles accordingly
            int* config = triTable[type];
            int e, e0, e1;
            for (int l = 0; l < 15; l++) {
                e = config[l];
                e0 = edge_map[e][0]; 
                e1 = edge_map[e][1];
                if (e != -1) {
                    geom[15 * idx + l] = (verts[e0] + verts[e1]) * (0.5f);
                    geom[15 * idx + l].w = 1.0f;
                }
                else { break; }
            }

        }
    }
}

void generate_data(float4* points, float4* grid, float4* geom, float4* color_black, float4* color_white)
{

    // Initialize points data.
    float delta = (xmax - xmin) / (numPoints - 1);
    for (int i = 0; i < numPoints; i++) {
        for (int j = 0; j < numPoints; j++) {
            for (int k = 0; k < numPoints; k++) {

                int idx = i + j * numPoints + k * numPoints * numPoints;

                // Set initial position data
                points[idx].x = xmin + delta * i;
                points[idx].y = xmax - delta * j;
                points[idx].z = xmin + delta * k;
                points[idx].w = 1.0f;

                color_white[idx].x = 1.0f;
                color_white[idx].y = 1.0f;
                color_white[idx].z = 1.0f;
                color_white[idx].w = 1.0f;

                color_black[idx].x = 0.0f;
                color_black[idx].y = 0.0f;
                color_black[idx].z = 0.0f;
                color_black[idx].w = 1.0f;
            }
        }
    }
    
    // Initialize grid data.
    for (int i = 0; i < (numPoints - 1); i++) {
        for (int j = 0; j < (numPoints - 1); j++) {
            for (int k = 0; k < (numPoints - 1); k++) {

                int idx_pt = i + j * numPoints + k * numPoints * numPoints;
                int idx_sq = idx_pt - j + k - 2 * k * numPoints;

                // Set initial position data
                grid[16 * idx_sq + 0] = points[idx_pt];
                grid[16 * idx_sq + 1] = points[idx_pt + 1];
                grid[16 * idx_sq + 2] = points[idx_pt + numPoints + 1];
                grid[16 * idx_sq + 3] = points[idx_pt + numPoints];

                grid[16 * idx_sq + 4] = points[idx_pt + numPoints * numPoints];
                grid[16 * idx_sq + 5] = points[idx_pt + numPoints * numPoints + 1];
                grid[16 * idx_sq + 6] = points[idx_pt + numPoints * numPoints + numPoints + 1];
                grid[16 * idx_sq + 7] = points[idx_pt + numPoints * numPoints + numPoints];

                grid[16 * idx_sq + 8] = points[idx_pt];
                grid[16 * idx_sq + 9] = points[idx_pt + 1];
                grid[16 * idx_sq + 10] = points[idx_pt + numPoints * numPoints + 1];
                grid[16 * idx_sq + 11] = points[idx_pt + numPoints * numPoints];

                grid[16 * idx_sq + 12] = points[idx_pt + numPoints];
                grid[16 * idx_sq + 13] = points[idx_pt + numPoints + 1];
                grid[16 * idx_sq + 14] = points[idx_pt + numPoints * numPoints + numPoints + 1];
                grid[16 * idx_sq + 15] = points[idx_pt + numPoints * numPoints + numPoints];

            }
        }
    }
    
    // Initialize geometry data.
    float4 zero = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    for (int k = 0; k < (numPoints - 1) * (numPoints - 1) * (numPoints - 1) * 15; k++) {
        geom[k] = zero;
    }
}

// Create VBOs
void createVBOs(GLuint* vao, GLuint* vbo1, GLuint* vbo2)
{
    glGenVertexArrays(2, vao);
    glBindVertexArray(vao[0]);
    // Create vertex buffer
    glGenBuffers(4, vbo1);

    // Initialize points and grid
    unsigned int points_size;
    float4* points;
    unsigned int grid_size;
    float4* grid;
    unsigned int geom_size;
    float4* geom;
    unsigned int color_size;
    float4* color_black;
    float4* color_white;

    // Allocate memory
    points_size = numPoints * numPoints * numPoints * sizeof(float4);
    points = (float4*)malloc(points_size);
    grid_size = (numPoints - 1) * (numPoints - 1) * (numPoints - 1) * 16
        * sizeof(float4);
    grid = (float4*)malloc(grid_size);
    geom_size = (numPoints - 1) * (numPoints - 1) * (numPoints - 1) * 15
        * sizeof(float4);
    geom = (float4*)malloc(geom_size);
    color_size = numPoints * numPoints * numPoints * sizeof(float4);
    color_white = (float4*)malloc(color_size);
    color_black = (float4*)malloc(color_size);
    // Initialize data
    generate_data(points, grid, geom, color_black, color_white);
    
    /*
        VAO[0]
    */
    /*
        VBO1[0] - grid
    */
    // Activate VBO id to use.
    glBindBuffer(GL_ARRAY_BUFFER, vbo1[0]);
    // Upload data to video card.
    glBufferData(GL_ARRAY_BUFFER, grid_size, grid, GL_STATIC_DRAW);

    glEnableVertexAttribArray(0);
    glBindBuffer(GL_ARRAY_BUFFER, vbo1[0]);
    glVertexAttribPointer(0, 4, GL_FLOAT, GL_FALSE, 0, (void*)NULL);

    /*
        VBO1[1] - points formerly color
    */
    // Activate VBO id to use.
    glBindBuffer(GL_ARRAY_BUFFER, vbo1[1]);
    // Upload data to video card.
    glBufferData(GL_ARRAY_BUFFER, points_size, points, GL_STATIC_DRAW);

    glEnableVertexAttribArray(1);
    glBindBuffer(GL_ARRAY_BUFFER, vbo1[1]);
    glVertexAttribPointer(1, 4, GL_FLOAT, GL_FALSE, 0, (void*)NULL);

    // Register buffer objects with CUDA
    if (cudaGLRegisterBufferObject(vbo1[1]) != hipSuccess)
    {
        printf("Could not register vbo for CUDA access!\r\n");
    }

    /*
        VBO1[2] - geometry
    */
    // Activate VBO id to use.
    glBindBuffer(GL_ARRAY_BUFFER, vbo1[2]);
    // Upload data to video card.
    glBufferData(GL_ARRAY_BUFFER, geom_size, geom, GL_STATIC_DRAW);

    glEnableVertexAttribArray(2);
    glBindBuffer(GL_ARRAY_BUFFER, vbo1[2]);
    glVertexAttribPointer(2, 4, GL_FLOAT, GL_FALSE, 0, (void*)NULL);

    // Register buffer objects with CUDA
    if (cudaGLRegisterBufferObject(vbo1[2]) != hipSuccess)
    {
        printf("Could not register vbo for CUDA access!\r\n");
    }

    /*
        VBO1[3] - color_white
    */
    // Activate VBO id to use.
    glBindBuffer(GL_ARRAY_BUFFER, vbo1[3]);
    // Upload data to video card.
    glBufferData(GL_ARRAY_BUFFER, color_size, color_white, GL_STATIC_DRAW);

    glEnableVertexAttribArray(3);
    glBindBuffer(GL_ARRAY_BUFFER, vbo1[3]);
    glVertexAttribPointer(3, 4, GL_FLOAT, GL_FALSE, 0, (void*)NULL);

    /*
        VAO[1]
    */
    /*
        VBO2[0] - grid
    */
    glBindVertexArray(vao[1]);
    // Create vertex buffer
    glGenBuffers(4, vbo2);

    // Activate VBO id to use.
    glBindBuffer(GL_ARRAY_BUFFER, vbo2[0]);
    // Upload data to video card.
    glBufferData(GL_ARRAY_BUFFER, grid_size, grid, GL_STATIC_DRAW);

    glEnableVertexAttribArray(0);
    glBindBuffer(GL_ARRAY_BUFFER, vbo2[0]);
    glVertexAttribPointer(0, 4, GL_FLOAT, GL_FALSE, 0, (void*)NULL);

    /*
        VBO2[1] - points formerly color
    */
    // Activate VBO id to use.
    glBindBuffer(GL_ARRAY_BUFFER, vbo2[1]);
    // Upload data to video card.
    glBufferData(GL_ARRAY_BUFFER, points_size, points, GL_STATIC_DRAW);

    glEnableVertexAttribArray(1);
    glBindBuffer(GL_ARRAY_BUFFER, vbo2[1]);
    glVertexAttribPointer(1, 4, GL_FLOAT, GL_FALSE, 0, (void*)NULL);

    // Register buffer objects with CUDA
    if (cudaGLRegisterBufferObject(vbo2[1]) != hipSuccess)
    {
        printf("Could not register vbo for CUDA access!\r\n");
    }

    /*
        VBO2[2] - geometry
    */
    // Activate VBO id to use.
    glBindBuffer(GL_ARRAY_BUFFER, vbo2[2]);
    // Upload data to video card.
    glBufferData(GL_ARRAY_BUFFER, geom_size, geom, GL_STATIC_DRAW);

    glEnableVertexAttribArray(2);
    glBindBuffer(GL_ARRAY_BUFFER, vbo2[2]);
    glVertexAttribPointer(2, 4, GL_FLOAT, GL_FALSE, 0, (void*)NULL);

    // Register buffer objects with CUDA
    if (cudaGLRegisterBufferObject(vbo2[2]) != hipSuccess)
    {
        printf("Could not register vbo for CUDA access!\r\n");
    }

    /*
        VBO2[3] - color_black
    */
    // Activate VBO id to use.
    glBindBuffer(GL_ARRAY_BUFFER, vbo2[3]);
    // Upload data to video card.
    glBufferData(GL_ARRAY_BUFFER, color_size, color_black, GL_STATIC_DRAW);

    glEnableVertexAttribArray(3);
    glBindBuffer(GL_ARRAY_BUFFER, vbo2[3]);
    glVertexAttribPointer(4, 3, GL_FLOAT, GL_FALSE, 0, (void*)NULL);

    // Release VBOs with ID 0 after use.
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    glBindVertexArray(0);

    // Free temporary data
    free(points); free(grid); free(geom); free(color_black); free(color_white);

    // Execute the algorithm
    run_cuda_kernel(vao, vbo1, vbo2);
}

// Delete VBOs
void deleteVBOs(GLuint* vbo)
{
    // Delete VBOs
    glBindBuffer(1, vbo[0]);
    glDeleteBuffers(1, &vbo[0]);
    glBindBuffer(1, vbo[1]);
    glDeleteBuffers(1, &vbo[1]);
    glBindBuffer(1, vbo[2]);
    glDeleteBuffers(1, &vbo[2]);

    // Unregister buffer objects with CUDA
    if ((cudaGLUnregisterBufferObject(vbo[1]) | cudaGLUnregisterBufferObject(vbo[2])) != hipSuccess)
    {
        printf("Failure while unregistering vbos!\r\n");
    }

    // Free VBOs
    *vbo = 0;
}

int getNumPoints()
{
    return numPoints;
}

void setNumPoints(int n)
{
    numPoints = n;
}