#ifndef _MARCHING_CUBES_KERNEL_CU_
#define _MARCHING_CUBES_KERNEL_CU_

#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime_api.h>    // includes for helper CUDA functions
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime_api.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>



#endif